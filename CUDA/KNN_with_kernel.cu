#include "hip/hip_runtime.h"
/* KNN algorithm using thrust library on CUDA (version with Kernel)

Obs:
A knn file might contain:
1. title: Latitude, Longitude
2. N: Total number of reference points
3. K: Number for K nearest points
4. Coordinates of the query point (only one coordinate)
5. Coordinates of all reference points (size: N)

Example of file:
Latitude,Longitude
8
3
-12.054251,-77.099688
-12.078973,-77.093252
-12.067450,-77.078543
-12.043947,-77.098275
-12.058031,-77.073309
-12.029678,-77.082271
-12.068602,-77.075530
-12.036418,-77.096255
-12.060808,-77.126168
*/

#include <stdio.h>
#include <fstream>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>

using namespace std;

string input_path = "../../../Datasets/Knn/input/knn_64.txt";	// put your path file here

__constant__ __device__ double DEG_TO_RAD = 0.017453292519943295769;
__constant__ __device__ double EARTH_RADIUS_IN_METERS = 6372797.560856;	// in meters
__constant__ __device__ double query_point[2]; // It contains the latitude and longitude of the query point

template <typename T>
void find_K_N(T& N, T& K)
{
	ifstream coordinatesFile;
	coordinatesFile.open(input_path);

	string titleStr, NStr, kStr;
	if (coordinatesFile.is_open()) {
		getline(coordinatesFile, titleStr);

		getline(coordinatesFile, NStr);
		N = (T)atof(NStr.c_str());

		getline(coordinatesFile, kStr);
		K = (T)atof(kStr.c_str());
	}
}


/* This function reads the coordinates from a file and fill to two host vectors
@param lats: vector in host of latitudes
@param lons: vector in host of longitudes */
template <typename T>
void readCoordinates(T*& latitudes, T*& longitudes, T*& queryPoint, int N)
{
	ifstream coordinatesFile;
	coordinatesFile.open(input_path);

	string latitStr, longitStr, titleStr, NStr, kStr;

	if (coordinatesFile.is_open()) {
		getline(coordinatesFile, titleStr);
		getline(coordinatesFile, NStr);
		getline(coordinatesFile, kStr);

		// For the first point - query point
		getline(coordinatesFile, latitStr, ',');
		queryPoint[0] = (T)atof(latitStr.c_str());

		getline(coordinatesFile, longitStr);
		queryPoint[1] = (T)atof(longitStr.c_str());

		for (int i = 0; i < N; ++i) {
			getline(coordinatesFile, latitStr, ',');
			latitudes[i] = (T)atof(latitStr.c_str());
			getline(coordinatesFile, longitStr);
			longitudes[i] = (T)atof(longitStr.c_str());
		}
	}
}

// Find the K nearest coordinates
template <typename T>
void getKNearest(T*&  h_lats, T*&  h_lons, T*&  k_lats, T*&  k_lons, int*&  h_seq, int& k)
{
	for (size_t i = 0; i < k; ++i) {
		k_lats[i] = h_lats[h_seq[i]];
		k_lons[i] = h_lons[h_seq[i]];
	}
}


// With Haversine Formula
template <typename T>
__global__ void computeDistance(T* lats, T* lons, T* distances, int n)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < n) {
		T latitudeArc = (lats[idx] - query_point[0]) * DEG_TO_RAD;
		T longitudeArc = (lons[idx] - query_point[1]) * DEG_TO_RAD;
		T latitudeH = sin(latitudeArc * 0.5);
		latitudeH *= latitudeH;
		T lontitudeH = sin(longitudeArc * 0.5);
		lontitudeH *= lontitudeH;
		T tmp = cos(lats[idx] * DEG_TO_RAD) * cos(query_point[0] * DEG_TO_RAD);
		distances[idx] = EARTH_RADIUS_IN_METERS * 2.0 * asin(sqrt(latitudeH + tmp*lontitudeH));
	}
}

template <typename T>
void printResults(int N, int K, T*& h_nearest_lat, T*& h_nearest_lon)
{
	printf("N: %d \nK: %d \nK-NN coordinates: \n", N, K);
	for (int i = 0; i < K; ++i) printf("%.6f, %.6f \n", h_nearest_lat[i], h_nearest_lon[i]);
}

int main(void)
{
	// Find the total of points from a file
	int N, K;
	find_K_N(N, K);

	if (!N) {
		cerr << "Unable to open file" << endl;
		return 0;
	}

	// Util pointers to host (h_) and device (_d)
	double *h_latitudes, *h_longitudes, *h_queryPoint, *h_distances;
	int* h_sequence;
	double *d_latitudes, *d_longitudes, *d_distances;

	// Data for device
	const int TOTAL_SIZE_BYTES = N * sizeof(double);

	// Allocate CPU memory
	h_latitudes = (double*)malloc(TOTAL_SIZE_BYTES);
	h_longitudes = (double*)malloc(TOTAL_SIZE_BYTES);
	h_distances = (double*)malloc(TOTAL_SIZE_BYTES);
	h_sequence = (int*)malloc(TOTAL_SIZE_BYTES);
	h_queryPoint = (double*)malloc(2 * sizeof(double));

	// Allocate GPU memory
	hipMalloc((void**)&d_latitudes, TOTAL_SIZE_BYTES);
	hipMalloc((void**)&d_longitudes, TOTAL_SIZE_BYTES);
	hipMalloc((void**)&d_distances, TOTAL_SIZE_BYTES);

	// Fill data in host
	readCoordinates(h_latitudes, h_longitudes, h_queryPoint, N);

	// Transfer the array host to the array device
	hipMemcpy(d_latitudes, h_latitudes, TOTAL_SIZE_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_longitudes, h_longitudes, TOTAL_SIZE_BYTES, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(query_point), &h_queryPoint, 2 * sizeof(double));		// CONSTANT MEMORY

	// wrap raw pointer with a device_ptr 
	thrust::device_ptr<double> d_dist_ptr = thrust::device_pointer_cast(d_distances);

	// Fill distances in device with the Haversine Formula
	computeDistance << <1, N >> > (d_latitudes, d_longitudes, d_distances, N);

	// Create a sequence. It work as iterator (indices to latitudes and longitudes)
	thrust::device_ptr<int> d_sequence_ptr = thrust::device_malloc<int>(N);
	thrust::sequence(d_sequence_ptr, d_sequence_ptr + N);

	// Sort by distances and the sequence vector must be reordered
	thrust::sort_by_key(d_dist_ptr, d_dist_ptr + N, d_sequence_ptr, thrust::less<int>());

	// Get only first K latitudes and longitudes
	double *h_nearest_lat, *h_nearest_lon;
	h_nearest_lat = (double*)malloc(K * sizeof(double));
	h_nearest_lon = (double*)malloc(K * sizeof(double));

	// Transfer the array host to the array device
	hipMemcpy(h_sequence, thrust::raw_pointer_cast(d_sequence_ptr), N * sizeof(int), hipMemcpyDeviceToHost);

	// Find the K nearest latitudes and longitudes
	getKNearest(h_latitudes, h_longitudes, h_nearest_lat, h_nearest_lon, h_sequence, K);

	// Delete device memory
	hipFree(d_latitudes), hipFree(d_longitudes), hipFree(d_distances);

	// Only for print results
	printResults(N, K, h_nearest_lat, h_nearest_lon);

	return 0;
}